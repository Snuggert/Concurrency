#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>

#include "file.h"
#include "timer.h"

using namespace std;

typedef double(*func_t)(double x);

/*
 * Simple gauss with mu=0, sigma^1=1
 */
double gauss(double x)
{
    return exp((-1 * x * x) / 2);
}

void fill(float *array, int offset, int range, float sample_start,
        float sample_end, func_t f)
{
    int i;
    float dx;

    dx = (sample_end - sample_start) / range;
    for (i = 0; i < range; i++) {
        array[i + offset] = f(sample_start + i * dx);
    }
}

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


__global__ void vectorAddKernel(float* deviceA, float* deviceB, float* deviceResult) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    deviceResult[index] = deviceA[index] + deviceB[index];
}

__global__ void waveStep(int N, float* old, float* current, float* next){
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (i < (N - 1)){
        next[i] = (2.0 * current[i]) - old[i] +
            (0.2 * (current[i-1] - (2.0 * current[i]) + 
            current[i+1]));
    }
}

void simulateCuda(int n, int max_t, float* old, float* current, float* next){
    int threadBlockSize = 512;

    // allocate the vectors on the GPU
    float* deviceOld = NULL;
    checkCudaCall(hipMalloc((void **) &deviceOld, n * sizeof(float)));
    if (deviceOld == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }
    float* deviceCurrent = NULL;
    checkCudaCall(hipMalloc((void **) &deviceCurrent, n * sizeof(float)));
    if (deviceCurrent == NULL) {
        checkCudaCall(hipFree(deviceOld));
        cout << "could not allocate memory!" << endl;
        return;
    }
    float* deviceNext = NULL;
    checkCudaCall(hipMalloc((void **) &deviceNext, n * sizeof(float)));
    if (deviceNext == NULL) {
        checkCudaCall(hipFree(deviceOld));
        checkCudaCall(hipFree(deviceCurrent));
        cout << "could not allocate memory!" << endl;
        return;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for(int i = 0; i < max_t; i++){

        // copy the original vectors to the GPU
        checkCudaCall(hipMemcpy(deviceOld, old, n*sizeof(float), hipMemcpyHostToDevice));
        checkCudaCall(hipMemcpy(deviceCurrent, current, n*sizeof(float), hipMemcpyHostToDevice));

        // execute kernel
        hipEventRecord(start, 0);
        vectorAddKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceOld, deviceCurrent, deviceNext);
        hipEventRecord(stop, 0);

        // check whether the kernel invocation was successful
        checkCudaCall(hipGetLastError());

        /* Copy results to old and current */
        checkCudaCall(hipMemcpy(deviceOld, deviceCurrent, n*sizeof(float), hipMemcpyDeviceToDevice));
        checkCudaCall(hipMemcpy(deviceCurrent, deviceNext, n*sizeof(float), hipMemcpyDeviceToDevice));
    }
    // copy result back
    checkCudaCall(hipMemcpy(next, deviceNext, n * sizeof(float), hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(deviceOld));
    checkCudaCall(hipFree(deviceCurrent));
    checkCudaCall(hipFree(deviceNext));

    // print the time the kernel invocation took, without the copies!
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    cout << "kernel invocation took " << elapsedTime << " milliseconds" << endl;

}


void vectorAddCuda(int n, float* a, float* b, float* result) {
    int threadBlockSize = 512;

    // allocate the vectors on the GPU
    float* deviceA = NULL;
    checkCudaCall(hipMalloc((void **) &deviceA, n * sizeof(float)));
    if (deviceA == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }
    float* deviceB = NULL;
    checkCudaCall(hipMalloc((void **) &deviceB, n * sizeof(float)));
    if (deviceB == NULL) {
        checkCudaCall(hipFree(deviceA));
        cout << "could not allocate memory!" << endl;
        return;
    }
    float* deviceResult = NULL;
    checkCudaCall(hipMalloc((void **) &deviceResult, n * sizeof(float)));
    if (deviceResult == NULL) {
        checkCudaCall(hipFree(deviceA));
        checkCudaCall(hipFree(deviceB));
        cout << "could not allocate memory!" << endl;
        return;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // copy the original vectors to the GPU
    checkCudaCall(hipMemcpy(deviceA, a, n*sizeof(float), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceB, b, n*sizeof(float), hipMemcpyHostToDevice));

    // execute kernel
    hipEventRecord(start, 0);
    vectorAddKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceA, deviceB, deviceResult);
    hipEventRecord(stop, 0);

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    checkCudaCall(hipMemcpy(result, deviceResult, n * sizeof(float), hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(deviceA));
    checkCudaCall(hipFree(deviceB));
    checkCudaCall(hipFree(deviceResult));

    // print the time the kernel invocation took, without the copies!
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    cout << "kernel invocation took " << elapsedTime << " milliseconds" << endl;
}


int main(int argc, char* argv[]) {
    float *old, *current, *next;
    timer vectorAddTimer("vector add timer");
    int t_max, i_max, num_threads;

    /* Parse commandline args: i_max t_max num_threads */
    if (argc < 4) {
        printf("Usage: %s i_max t_max num_threads [initial_data]\n", argv[0]);
        printf(" - i_max: number of discrete amplitude points, should be >2\n");
        printf(" - t_max: number of discrete timesteps, should be >=1\n");
        printf(" - num_threads: number of threads to use for simulation, "
                "should be >=1\n");
        printf(" - initial_data: select what data should be used for the first "
                "two generation.\n");
        printf("   Available options are:\n");
        printf("    * sin: one period of the sinus function at the start.\n");
        printf("    * sinfull: entire data is filled with the sinus.\n");
        printf("    * gauss: a single gauss-function at the start.\n");
        printf("    * file <2 filenames>: allows you to specify a file with on "
                "each line a float for both generations.\n");

        exit(1);
    }

    i_max = atoi(argv[1]);
    t_max = atoi(argv[2]);
    num_threads = atoi(argv[3]);

    if (i_max < 3) {
        printf("argument error: i_max should be >2.\n");
        exit(1);
    }
    if (t_max < 1) {
        printf("argument error: t_max should be >=1.\n");
        exit(1);
    }
    if (num_threads < 1) {
        printf("argument error: num_threads should be >=1.\n");
        exit(1);
    }

    /* Allocate and initialize buffers. */
    old = new float[i_max];
    current = new float[i_max];
    next = new float[i_max];

    if (old == NULL || current == NULL || next == NULL) {
        fprintf(stderr, "Could not allocate enough memory, aborting.\n");
        exit(1);
    }

    memset(old, 0, i_max * sizeof(float));
    memset(current, 0, i_max * sizeof(float));
    memset(next, 0, i_max * sizeof(float));

    /* How should we will our first two generations? */
    if (argc > 4) {
        if (strcmp(argv[4], "sin") == 0) {
            fill(old, 1, i_max/4, 0, 2*3.14, sin);
            fill(current, 2, i_max/4, 0, 2*3.14, sin);
        } else if (strcmp(argv[4], "sinfull") == 0) {
            fill(old, 1, i_max-2, 0, 10*3.14, sin);
            fill(current, 2, i_max-3, 0, 10*3.14, sin);
        } else if (strcmp(argv[4], "gauss") == 0) {
            fill(old, 1, i_max/4, -3, 3, gauss);
            fill(current, 2, i_max/4, -3, 3, gauss);
        } else if (strcmp(argv[4], "file") == 0) {
            if (argc < 7) {
                printf("No files specified!\n");
                exit(1);
            }
            file_read_float_array(argv[5], old, i_max);
            file_read_float_array(argv[6], current, i_max);
        } else {
            printf("Unknown initial mode: %s.\n", argv[4]);
            exit(1);
        }
    } else {
        /* Default to sinus. */
        fill(old, 1, i_max/4, 0, 2*3.14, sin);
        fill(current, 2, i_max/4, 0, 2*3.14, sin);
    }

    vectorAddTimer.start();
    vectorAddTimer.stop();

    cout << vectorAddTimer;
    cout << "results OK!" << endl;
            
    delete[] old;
    delete[] current;
    delete[] next;
    
    return 0;
}
