#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>

#include "file.h"
#include "timer.h"

using namespace std;

typedef double(*func_t)(double x);

/*
 * Simple gauss with mu=0, sigma^1=1
 */
double gauss(double x)
{
    return exp((-1 * x * x) / 2);
}

void fill(float *array, int offset, int range, float sample_start,
        float sample_end, func_t f)
{
    int i;
    float dx;

    dx = (sample_end - sample_start) / range;
    for (i = 0; i < range; i++) {
        array[i + offset] = f(sample_start + i * dx);
    }
}

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


__global__ void vectorAddKernel(float* deviceA, float* deviceB, float* deviceResult) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    deviceResult[index] = deviceA[index] + deviceB[index];
}


void vectorAddCuda(int n, float* a, float* b, float* result) {
    int threadBlockSize = 512;

    // allocate the vectors on the GPU
    float* deviceA = NULL;
    checkCudaCall(hipMalloc((void **) &deviceA, n * sizeof(float)));
    if (deviceA == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }
    float* deviceB = NULL;
    checkCudaCall(hipMalloc((void **) &deviceB, n * sizeof(float)));
    if (deviceB == NULL) {
        checkCudaCall(hipFree(deviceA));
        cout << "could not allocate memory!" << endl;
        return;
    }
    float* deviceResult = NULL;
    checkCudaCall(hipMalloc((void **) &deviceResult, n * sizeof(float)));
    if (deviceResult == NULL) {
        checkCudaCall(hipFree(deviceA));
        checkCudaCall(hipFree(deviceB));
        cout << "could not allocate memory!" << endl;
        return;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // copy the original vectors to the GPU
    checkCudaCall(hipMemcpy(deviceA, a, n*sizeof(float), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceB, b, n*sizeof(float), hipMemcpyHostToDevice));

    // execute kernel
    hipEventRecord(start, 0);
    vectorAddKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceA, deviceB, deviceResult);
    hipEventRecord(stop, 0);

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    checkCudaCall(hipMemcpy(result, deviceResult, n * sizeof(float), hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(deviceA));
    checkCudaCall(hipFree(deviceB));
    checkCudaCall(hipFree(deviceResult));

    // print the time the kernel invocation took, without the copies!
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    cout << "kernel invocation took " << elapsedTime << " milliseconds" << endl;
}


int main(int argc, char* argv[]) {
    float *old, *current, *next;
    int t_max, i_max, num_threads;
    double time;

    /* Parse commandline args: i_max t_max num_threads */
    if (argc < 4) {
        printf("Usage: %s i_max t_max num_threads [initial_data]\n", argv[0]);
        printf(" - i_max: number of discrete amplitude points, should be >2\n");
        printf(" - t_max: number of discrete timesteps, should be >=1\n");
        printf(" - num_threads: number of threads to use for simulation, "
                "should be >=1\n");
        printf(" - initial_data: select what data should be used for the first "
                "two generation.\n");
        printf("   Available options are:\n");
        printf("    * sin: one period of the sinus function at the start.\n");
        printf("    * sinfull: entire data is filled with the sinus.\n");
        printf("    * gauss: a single gauss-function at the start.\n");
        printf("    * file <2 filenames>: allows you to specify a file with on "
                "each line a float for both generations.\n");

        exit(1);
    }

    i_max = atoi(argv[1]);
    t_max = atoi(argv[2]);
    num_threads = atoi(argv[3]);

    if (i_max < 3) {
        printf("argument error: i_max should be >2.\n");
        exit(1);
    }
    if (t_max < 1) {
        printf("argument error: t_max should be >=1.\n");
        exit(1);
    }
    if (num_threads < 1) {
        printf("argument error: num_threads should be >=1.\n");
        exit(1);
    }

    /* Allocate and initialize buffers. */
    old = malloc(i_max * sizeof(float));
    current = malloc(i_max * sizeof(float));
    next = malloc(i_max * sizeof(float));

    if (old == NULL || current == NULL || next == NULL) {
        fprintf(stderr, "Could not allocate enough memory, aborting.\n");
        exit(1);
    }

    memset(old, 0, i_max * sizeof(float));
    memset(current, 0, i_max * sizeof(float));
    memset(next, 0, i_max * sizeof(float));

    /* How should we will our first two generations? */
    if (argc > 4) {
        if (strcmp(argv[4], "sin") == 0) {
            fill(old, 1, i_max/4, 0, 2*3.14, sin);
            fill(current, 2, i_max/4, 0, 2*3.14, sin);
        } else if (strcmp(argv[4], "sinfull") == 0) {
            fill(old, 1, i_max-2, 0, 10*3.14, sin);
            fill(current, 2, i_max-3, 0, 10*3.14, sin);
        } else if (strcmp(argv[4], "gauss") == 0) {
            fill(old, 1, i_max/4, -3, 3, gauss);
            fill(current, 2, i_max/4, -3, 3, gauss);
        } else if (strcmp(argv[4], "file") == 0) {
            if (argc < 7) {
                printf("No files specified!\n");
                exit(1);
            }
            file_read_double_array(argv[5], old, i_max);
            file_read_double_array(argv[6], current, i_max);
        } else {
            printf("Unknown initial mode: %s.\n", argv[4]);
            exit(1);
        }
    } else {
        /* Default to sinus. */
        fill(old, 1, i_max/4, 0, 2*3.14, sin);
        fill(current, 2, i_max/4, 0, 2*3.14, sin);
    }

    vectorAddTimer.start();
    vectorAddTimer.stop();

    cout << vectorAddTimer;
    cout << "results OK!" << endl;
            
    delete[] old;
    delete[] current;
    delete[] next;
    
    return 0;
}
