#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include <time.h>

#include "timer.h"

using namespace std;

typedef double(*func_t)(double x);

/*
 * Simple gauss with mu=0, sigma^1=1
 */
double gauss(double x)
{
    return exp((-1 * x * x) / 2);
}

void fillRandom(float *array, int n)
{
    int i;
    srand(time(NULL));

    for (i = 0; i < n; i++) {
        array[i] = rand();
    }
}

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

__global__ void cudaMax(int N, int gap, float* values){
    unsigned i = (blockIdx.x * blockDim.x + threadIdx.x) * (2 * gap);
    if ((i + gap) < N){
        values[i] = (values[i] < values[i + gap]) ? values[i + gap] : values[i];
    }
}

void calculateMax(int n, float* values){
    int threadBlockSize = 512;
    int gap = 0;
    float max = 0;

    // allocate the vectors on the GPU
    float* deviceValues = NULL;
    checkCudaCall(hipMalloc((void **) &deviceValues, n * sizeof(float)));
    if (deviceValues == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    while(n != 1){
        n = n/2;

        // copy the original vectors to the GPU
        checkCudaCall(hipMemcpy(deviceValues, values, n*sizeof(float), hipMemcpyHostToDevice));

        // execute kernel
        hipEventRecord(start, 0);
        cudaMax<<<ceilf((float) n/threadBlockSize), threadBlockSize>>>(n, gap, deviceValues);
        hipEventRecord(stop, 0);
        // check whether the kernel invocation was successful
        checkCudaCall(hipGetLastError());

        gap = gap * 2; 

        /* Copy results to old and current */
    }
    // copy result back
    checkCudaCall(hipMemcpy(values, deviceValues, n * sizeof(float), hipMemcpyDeviceToHost));
    max = values[0];
    cout << max << endl;

    checkCudaCall(hipFree(deviceValues));

    // print the time the kernel invocation took, without the copies!
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    cout << "kernel invocation took " << elapsedTime << " milliseconds" << endl;

}

int main(int argc, char* argv[]) {
    float* values;
    timer vectorAddTimer("vector add timer");
    int n = 4000;

    values = new float[n];
    memset(values, 0, n * sizeof(float));
    fillRandom(values, n);

    vectorAddTimer.start();
    calculateMax(n, values);
    vectorAddTimer.stop();

    cout << vectorAddTimer;
    cout << "results OK!" << endl;
            
    delete[] values;
    
    return 0;
}
